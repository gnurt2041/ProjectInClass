// nvcc -gencode arch=compute_50,code=sm_50 FFT_forward.cu -o FFT_forward
// ./FFT_forward

#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <chrono>


using namespace std::chrono;
using namespace std;

typedef complex<double> base;
typedef double2 Complex_my;
#define PI acos(-1.0)

/**
* Parallel Functions for performing various tasks
*/

static __device__ __host__ inline Complex_my Add(Complex_my A, Complex_my B)
{
    Complex_my C;
    C.x = A.x + B.x;
    C.y = A.y + B.y;
    return C;
}

/**
 *  Inverse of Complex_my Number
 */
static __device__ __host__ inline Complex_my Inverse(Complex_my A)
{
    Complex_my C;
    C.x = -A.x;
    C.y = -A.y;
    return C;
}

/**
 *  Multipication of Complex_my Numbers
 */
static __device__ __host__ inline Complex_my Multiply(Complex_my A, Complex_my B)
{
    Complex_my C;
    C.x = A.x * B.x - A.y * B.y;
    C.y = A.y * B.x + A.x * B.y;
    return C;
}

/**
* Reorders array by bit-reversing the indexes.
*/
__global__ void bitrev_reorder(Complex_my *__restrict__ r, Complex_my *__restrict__ d, long s, size_t nthr, long n)
{
    long id = blockIdx.x * nthr + threadIdx.x;
    //r[id].x = -1;
    if (id < n and __brev(id) >> (32 - s) < n)
        r[__brev(id) >> (32 - s)] = d[id];
}

/**
* Inner part of the for loop
*/
__device__ void inplace_fft_inner(Complex_my *__restrict__ A, long i, long j, long len, long n)
{
    if (i + j + len / 2 < n and j < len / 2)
    {
        Complex_my u, v;

        float angle = (2 * M_PI * j) / len;
        v.x = cos(angle);
        v.y = -sin(angle);

        u = A[i + j];
        v = Multiply(A[i + j + len / 2], v);
        // prlongf("i:%d j:%d u_x:%f u_y:%f    v_x:%f v_y:%f\n", i, j, u.x, u.y, v.x, v.y);
        A[i + j] = Add(u, v);
        A[i + j + len / 2] = Add(u, Inverse(v));
    }
}

/**
* FFT if number of threads are sufficient.
*/
__global__ void inplace_fft(Complex_my *__restrict__ A, long i, long len, long n, long threads)
{
    long j = blockIdx.x * threads + threadIdx.x;
    inplace_fft_inner(A, i, j, len, n);
}

/**
* FFt if number of threads are not sufficient.
*/
__global__ void inplace_fft_outer(Complex_my *__restrict__ A, long len, long n, long threads)
{
    long i = (blockIdx.x * threads + threadIdx.x)*len;
    for (long j = 0; j < len / 2; j++)
    {
        inplace_fft_inner(A, i, j, len, n);
    }
}
/**
* parallel FFT transform and inverse transform
* Arguments vector of complex numbers, invert, balance, number of threads
* Perform inplace transform
*/
void fft(vector<base> &a, bool invert, long balance = 10, long threads = 32)
{
    // Creating array from vector
    long n = (long)a.size();
    long data_size = n * sizeof(Complex_my);
    Complex_my *data_array = (Complex_my *)malloc(data_size);
    for (long i = 0; i < n; i++)
    {
        data_array[i].x = a[i].real();
        data_array[i].y = a[i].imag();
    }

    // Copying data to GPU
    Complex_my *A, *dn;
    hipMalloc((void **)&A, data_size);
    hipMalloc((void **)&dn, data_size);
    hipMemcpy(dn, data_array, data_size, hipMemcpyHostToDevice);
    // Bit reversal reordering
    long s = log2(n);

    bitrev_reorder<<<ceil(float(n) / threads), threads>>>(A, dn, s, threads, n);

    // Synchronize
    hipDeviceSynchronize();
    // Iterative FFT with loop parallelism balancing
    for (long len = 2; len <= n; len <<= 1)
    {
        if (n / len > balance)
        {

            inplace_fft_outer<<<ceil((float)n / threads / len), threads>>>(A, len, n, threads);
        }
        else
        {
            for (long i = 0; i < n; i += len)
            {
                float repeats = len / 2;
                inplace_fft<<<ceil(repeats / threads), threads>>>(A, i, len, n, threads);
            }
        }
    }

    // Copy data from GPU
    Complex_my *result;
    result = (Complex_my *)malloc(data_size);
    hipMemcpy(result, A, data_size, hipMemcpyDeviceToHost);

    // Saving data to vector<complex> in input.
    for (long i = 0; i < n; i++)
    {
        a[i] = base(result[i].x, result[i].y);
    }
    // Free the memory blocks
    free(data_array);
    hipFree(A);
    hipFree(dn);
    return;
}


void fft_slow(vector<base> & a) // biến đổi de quy fft của vector a
{
    long n = a.size();
    if(n == 1)
    {
        return;
    }
    long i;
    vector<base>  a_even(n/2), a_odd (n/2);

    for (long i = 0; 2 * i < n; i++) {
        a_even[i] = a[2*i];
        a_odd[i] = a[2*i+1];
    }

    fft_slow(a_even);
    fft_slow(a_odd);

    for(i = 0; i < n / 2; i++)
    {
        base t = exp(base(0, -2 * PI * i / n)) * a_odd[i];
		a[i]  = a_even[i] + t;
		a[i + n / 2] = a_even[i] - t;
    }
}

double compare(vector<base>& v1, vector<base>& v2) {

    // Calculate the squared distances between all pairs
    vector<double> squaredDistances;
    for (long i = 0; i < v1.size(); i++) {
        base diff = v1[i] - v2[i];
        // double squaredDistance = diff.real() * diff.real() + diff.imag() * diff.imag();
        double squaredDistance  = round(abs(diff)/abs(v2[i]));
        squaredDistances.push_back(squaredDistance);
    }

    // Find the maximum squared distance
    double maxSquaredDistance = *max_element(squaredDistances.begin(), squaredDistances.end());

    // Calculate and return the maximum distance
    // return sqrt(maxSquaredDistance);
    return maxSquaredDistance;
}

// #define N 100000
// #define BALANCE 2

int main()
{
    long min_power, max_power;
    long BALANCE = 1024;
    string write_to_file;
    string save_csv;
    cout << "Enter min-max power of 2 (min -> max): ";
    cin >> min_power >> max_power;
    cout << "Write to file (y/n): ";
    cin >> write_to_file;

    cout << "Save to csv (y/n): ";
    cin >> save_csv;
    
    if(write_to_file == "y" || write_to_file == "Y"){
        freopen("output.txt", "w", stdout);
    }

    for (long power = min_power; power <= max_power; power+=1)
    {
        long N = pow(2, power);
        ofstream result(to_string(N) + "result.csv");
        result << "Cores,Parallel,Sequential,Speedup,Correct" << endl;
        cout << "Runing with N = " << N << endl;
        vector<long> a(N);
        for(long i = 0; i < N; i++)
        {
            a[i] = rand() % 100;
        }

        // For Sequential
        vector<base> fa_serial(a.begin(), a.end());
        auto begin = chrono::steady_clock::now();
        fft_slow(fa_serial);
        auto end = chrono::steady_clock::now();
        double time_fft_serial = chrono::duration_cast<chrono::microseconds>(end - begin).count();
        cout << "# of Cores    Parallel             Sequential       Speedup    Correct" << endl;
        long max_length = to_string(1024).length();
        for(long threads = 1; threads <= 1024; threads*=2)
        {
            /// For Parallel
            begin = chrono::steady_clock::now();
            vector<base> fa_paralel(a.begin(), a.end());
            fft(fa_paralel, BALANCE, threads);
            end = chrono::steady_clock::now();
            double time_fft_parallel = chrono::duration_cast<chrono::microseconds>(end - begin).count();

            string thread_str = to_string(threads);
            while(thread_str.length() < max_length){
                thread_str = ' ' + thread_str;
            }

            string correct;
            if(round(compare(fa_paralel,fa_serial)) == 0){
                correct = "True";
            }
            else correct = "False";
            cout << "   " <<  thread_str << "      " << scientific << time_fft_parallel << "(µs)" << "   " << scientific << time_fft_serial << "(µs)"
                << "   " << fixed << time_fft_serial/ time_fft_parallel
                << "     " << correct << endl;
            // cout << endl;
            if(save_csv == "y" || save_csv == "Y"){
                result << threads << "," << time_fft_parallel << "," << time_fft_serial << "," << time_fft_serial/ time_fft_parallel << "," << correct << endl;
            }
            if(correct == "False")
            {
                for(long i = 0; i < N; i++){
                    base diff = fa_paralel[i] - fa_serial[i];
                    cout << fa_paralel[i]  << " | " << fa_serial[i] << " -> " << round(abs(diff)/abs(fa_serial[i])) << endl;
                }
            }

        }
    }
    return 0;
}